#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <string>
#include <stdio.h>  
#include <stdlib.h>
#include <float.h>
#include <memory.h>
#include <math.h>
#include <stdbool.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cblas.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>
#include "expm.h"

// *** Matrix exponential program acting on matrices of type : [Double Complex] - 01/03/2019 ***


// TIME BREAKDOWN BY COMPUTE OPERATION - FROM PROFILER:
// 98.4% of runtime is spent in the multiply kernel
// 1.6% of runtime is spent in the inverse kernels

// Memory copies take small relative to operation time to complete for larger matrices
// Small overlap between dgemm kernels due to using large number of available resources so only a small
// amount of kernel concurrency possible
// For smaller matrices (e.g. 50 * 50) more concurrency is witnissed (50% overlap) - 'A more concurrent
// execution.'







void matrix_complex_print(hipDoubleComplex* A, int network_size){
	for (int j = 0; j < network_size; j++){
		printf("[");
		for (int k = 0; k < network_size; k++){
			printf(" %.15lf ", A[(j*network_size) + k].x );
			printf("+");
			printf(" %.15lfi ", A[(j*network_size) + k].y );
		}
		printf("]");
		printf("\n");
	}
}

void matrix_Square(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex *d_C, int n, double* multiply_total_time, int s){
	clock_t multiply_begin = clock();

	const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;
	
	hipMemcpy(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice);	// Copy first operand to the device (only one copy is needed for the squaring phase)

	for (int k = 0; k < s; k++) {
		hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_A, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication
        hipMemcpy(d_A, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);	// Copy product back to host
 	
 	clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;

}


void matrix_Multiply(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex* d_B, hipDoubleComplex *d_C, int n, double* multiply_total_time){

    clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    hipMemcpy(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice);	// Copy first operand to the device
    hipMemcpy(d_B, B, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice); // Copy second operand to the device

    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_B, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication

    hipMemcpy(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);	// Copy product back to host
    
    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;
}


void matrix_Multiply_With_Streams(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex* d_B, hipDoubleComplex *d_C, int n, double* multiply_total_time, hipStream_t stream){
 clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    hipMemcpyAsync(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice, stream); // Copy first operand to the device
    hipMemcpyAsync(d_B, B, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice, stream); // Copy second operand to the device

    hipblasSetStream(handle, stream);
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_B, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication

    hipMemcpyAsync(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost, stream); // Copy product back to host
    
    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;

}


hipDoubleComplex **get_Matrix_Powers_New(hipDoubleComplex *A, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, hipblasHandle_t handle, int n, double* multiply_total_time) {

    hipDoubleComplex **Tpowers = (hipDoubleComplex **) malloc(11 * sizeof(hipDoubleComplex *));

    for (int i = 0; i < 11; i++) {
        Tpowers[i] = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    }

    memcpy(Tpowers[1], A, n * n * sizeof(hipDoubleComplex));
    

    // To calculate Tpoers[2] & Tpowers[4] this is sequential
    matrix_Multiply(handle, Tpowers[1], Tpowers[1], Tpowers[2], d_A, d_B, d_C, n, multiply_total_time);
     
    matrix_Multiply(handle, Tpowers[2], Tpowers[2], Tpowers[4], d_A, d_B, d_C, n, multiply_total_time);



    matrix_Multiply(handle, Tpowers[4], Tpowers[2], Tpowers[6], d_A, d_B, d_C, n, multiply_total_time);
   
    matrix_Multiply(handle, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time);

    return Tpowers;
}


void InverseOfMatrix_Alternative_Two(hipDoubleComplex* L, hipDoubleComplex* inverse, int n, hipDoubleComplex* b){ // Calculate matrix inverse through LU factorisation
    
    hipsolverStatus_t  status;	// Link to the cusolver context
    hipsolverHandle_t handler;
    status = hipsolverDnCreate(&handler);

    hipDoubleComplex* A;
    int* dLUPivots_ALT;
    int* dLUInfo_ALT;
    hipDoubleComplex *buffer = NULL;
    int bufferSize = 0;
    int h_info = 0;
    hipDoubleComplex *x;

    hipMalloc(&A, sizeof(hipDoubleComplex)*n*n), "Failed to allocate A!";
    hipMalloc(&x, n * n*sizeof(hipDoubleComplex)), "Failed to allocate x!";
     
    hipMalloc(&dLUPivots_ALT, n * sizeof(int)), "Failed to allocate dLUPivots!";
    hipMalloc(&dLUInfo_ALT, sizeof(int)), "Failed to allocate dLUInfo!";
    hipMemcpy(A, L, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice), "Failed to copy to adL!";
    hipMemcpy(x, b, sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice);

    hipsolverDnZgetrf_bufferSize(handler, n, n, (hipDoubleComplex*)A, n, &bufferSize);
    hipMalloc(&buffer, sizeof(hipDoubleComplex)*bufferSize);
  
    status = hipsolverDnZgetrf(handler, n, n, A, n, buffer, dLUPivots_ALT, dLUInfo_ALT);
    if(status!=HIPSOLVER_STATUS_SUCCESS){
        printf("ERROR!!\n");
    } 

    hipMemcpy(&h_info, dLUInfo_ALT, sizeof(int), hipMemcpyDeviceToHost);
 
    if ( h_info != 0 ){
        fprintf(stderr, "Error: LU factorization failed\n");
        printf("%d\n", h_info );
    }
      
    hipsolverDnZgetrs(handler, HIPBLAS_OP_N, n, n, A, n, dLUPivots_ALT, x, n, dLUInfo_ALT);
    hipDeviceSynchronize();
     if(status!=HIPSOLVER_STATUS_SUCCESS){
        printf("ERROR!!\n");
    } 
    hipMemcpy(&h_info, dLUInfo_ALT, sizeof(int), hipMemcpyDeviceToHost);
        if ( h_info != 0 ){
        fprintf(stderr, "Error: LU factorization failed\n");
    }
    hipMemcpy(inverse, x, sizeof(hipDoubleComplex) * n * n, hipMemcpyDeviceToHost), "Failed to copy to res!";

    // Free device memory:
    hipFree(dLUPivots_ALT);
    hipFree(dLUInfo_ALT);
    hipFree(A);
    hipFree(x);
    hipFree(buffer);

}


void matrix_Subtract_New(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n) { // PARALLEL CANDIDATE

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            c[(n * i) + j] = hipCsub(a[(n * i) + j], b[(n * i) + j]); // Complex subtraction
        }
    }
}


void matrixAdd_New(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n) { // PARALLEL CANDIDATE

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            c[(n * i) + j] = hipCadd(a[(n * i) + j], b[(n * i) + j]); // Complex addition
        }
    }
}


void set_Identity_New(hipDoubleComplex *i_matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                i_matrix[(n * i) + j].x = 1;
            } else {
                i_matrix[(n * i) + j].x = 0;
            }
        }
    }
}


// *** CURRENT WORK ***
void scale_tester(hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, const hipDoubleComplex alf, int n){

    //const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    //hipblasZdgmm(handle, mode, n,n, d_A, n, d_X, n, d_C, n);
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, alpha, d_A, n, beta, NULL, n, d_C, n);
}


void matrix_scale_kernel(hipblasHandle_t &handle, hipDoubleComplex* A, hipDoubleComplex scale, int n){

	const hipDoubleComplex alf = make_hipDoubleComplex(scale.x, scale.y);
  
    const hipDoubleComplex *alpha = &alf;

	hipblasZscal(handle,n*n,alpha,A,1);
}

void matrix_Scale_New(hipDoubleComplex *a, hipDoubleComplex *scaled, hipDoubleComplex scale, int n, double* scale_total_time ) {
    clock_t scale_begin = clock();

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            //scaled[(n * i) + j] = a[(n * i) + j] * scale;
            scaled[(n * i) + j] = hipCmul(a[(n * i) + j],scale); // Complex multiplication
        }
    }

    clock_t scale_end = clock();
    double time_spent = (double)(scale_end - scale_begin) / CLOCKS_PER_SEC;
    scale_total_time[0] = scale_total_time[0] + time_spent;
}


void matrix_Absolute_New(hipDoubleComplex *a, hipDoubleComplex *b, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            b[(n * i) + j].x = hipCabs((a[(n * i) + j]));
            b[(n * i) + j].y = 0;
        }
    }
}


double calculate_one_norm_New_complex(const hipDoubleComplex *A, int n) {
    double max = -DBL_MAX;
	double count;
    for (int i = 0; i < n; i++) {
        count = 0;
        for (int j = 0; j < n; j++) {
            count += hipCabs((A[(n * j) + i]));
        }
        if (count > max) {;
            max = count;
        };
    }
    return max;
}


// COMPUTING OPTIMAL PARAMETERS
double ell(hipDoubleComplex *A, hipDoubleComplex *temp_new, double coeff, int m_val, int n, double* scale_total_time) {

    double norm_one, norm_two, p, alpha, output;
    memcpy(A, temp_new, n * n * sizeof(hipDoubleComplex));
    
    matrix_Absolute_New(A, temp_new, n);

    p = pow(coeff, (1.0 / (2 * m_val + 1)));
    
    matrix_Scale_New(temp_new, temp_new, make_hipDoubleComplex(p, 0), n, scale_total_time);
    
    norm_one = calculate_one_norm_New_complex(A, n);
    norm_two = calculate_one_norm_New_complex(temp_new, n);
    
    alpha = norm_two / norm_one;

    output = fmax(ceil(log2((2 * alpha) / 2.220446049250313e-16) / (2 * m_val)), 0);

    return output;
}

void get_pade_coefficients(double *buf, int m) {

    double coefficients[5][14] = {
            {120, 60, 12, 1},
            {30240, 15120, 3360, 420, 30, 1},
            {17297280, 8648640, 1995840, 277200, 25200, 1512, 56 ,1},
            {17643225600, 8821612800, 2075673600, 302702400, 30270240, 2162160, 110880, 3960, 90, 1},
            {64764752532480000, 32382376266240000, 7771770303897600, 1187353796428800, 129060195264000, 10559470521600, 670442572800, 33522128640, 1323241920, 40840800, 960960, 16380, 182, 1}
        };

    switch (m) {

        case 3  : {
            buf = coefficients[0];

        }
        case 5  : {
            buf = coefficients[1];
        }
        case 7  : {
            buf = coefficients[2];
        }

        case 9  : {
            buf = coefficients[3];
        }
        case 13  : {
            for (int i = 0; i < sizeof(coefficients[4]) / sizeof(double); i++) {
                buf[i] = coefficients[4][i];
            }
        }
        default:
            break;
    }
}

int main(){

    hipDoubleComplex* A;
    int n = 4000;

    // Allocate the pinned memory:
    hipHostMalloc((void**)&A, n*n*sizeof(hipDoubleComplex));


    // Initialize the pinned memory:
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; ++j)
        {
           A[(n*i) + j].x = (n*i) + j;
           A[(n*i) + j].y = (n*i) + j;
        }
    }
    

    // VARIABLES TO HOLD TOTAL COMPONENT TIMES:
    double* scale_total_time = (double *) malloc(1 * sizeof(double));
    double* multiply_total_time = (double *) malloc(1 * sizeof(double));

    clock_t begin = clock();  // Begin recording expm execution

    // CUBLAS HANDLE:
    hipblasHandle_t handle, handle2;
    hipblasCreate(&handle);
    hipblasCreate(&handle2);

    // Allocate 3 arrays on GPU
    hipDoubleComplex *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(hipDoubleComplex));
    hipMalloc(&d_B, n * n * sizeof(hipDoubleComplex));
    hipMalloc(&d_C, n * n * sizeof(hipDoubleComplex));


    double theta[5] = {1.495585217958292e-002, 2.539398330063230e-001,
                       9.504178996162932e-001, 2.097847961257068e+000,
                       5.371920351148152e+000};

    double error_coefficients[5] = {1 / 100800.0, 1 / 10059033600.0, 1 / 4487938430976000.0,
                                    1 / 113250775606021113483283660800000000.0,
                                    1 / 113250775606021113483283660800000000.0};

    // Allocate temporary arrays to hold temporary matrices used at various stages in the calculation
    hipDoubleComplex *identity_new;
    hipDoubleComplex *U_new;
    hipDoubleComplex *V_new;
    hipDoubleComplex *temp_new;
    hipDoubleComplex *temp_2_new;

    //identity_new = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&identity_new, n * n * sizeof(hipDoubleComplex));
    
    //U_new = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&U_new, n * n * sizeof(hipDoubleComplex));

    //V_new = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&V_new, n * n * sizeof(hipDoubleComplex));

    //temp_new = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&temp_new, n * n * sizeof(hipDoubleComplex));

    //temp_2_new = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&temp_2_new, n * n * sizeof(hipDoubleComplex));
    // LARGE MEMORY ALLOCATION PHASE


    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    float* A_d;
    hipMalloc(&A_d, n*n*sizeof(hipDoubleComplex));

    double d4, d6, d8, d10, eta1, eta3, eta4, eta5, s;

    ///////////////////////////////////////////////////////////////////////////////////////////////



    hipDoubleComplex **Tpowers;
    // = (hipDoubleComplex **) malloc(11 * sizeof(hipDoubleComplex *));
    hipHostMalloc((void**)&Tpowers, 11 * sizeof(hipDoubleComplex*));

    for (int i = 0; i < 11; i++) {
       // Tpowers[i] = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
        hipHostMalloc((void**)&Tpowers[i], n *n* sizeof(hipDoubleComplex));

    }

    memcpy(Tpowers[1], A, n * n * sizeof(hipDoubleComplex));

    // To calculate Tpowers[2] & Tpowers[4] this is sequential
    matrix_Multiply(handle, Tpowers[1], Tpowers[1], Tpowers[2], d_A, d_B, d_C, n, multiply_total_time);
     
    matrix_Multiply(handle, Tpowers[2], Tpowers[2], Tpowers[4], d_A, d_B, d_C, n, multiply_total_time);

    // Attemp to have concurrency:

    matrix_Multiply_With_Streams(handle, Tpowers[4], Tpowers[2], Tpowers[6], d_A, d_B, d_C, n, multiply_total_time, stream1);

    matrix_Multiply_With_Streams(handle2, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time, stream2);
    
   


    //matrix_Multiply_With_Streams(handle2, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time, stream2);
    ///////////////////////////////////////////////////////////////////////////////////////////////


    //hipDoubleComplex **Tpowers = get_Matrix_Powers_New(A, d_A, d_B, d_C, handle, n, multiply_total_time);
    
  
    // We know we need to calculate d4, d6 d8, d10 - they operate on independent data:



    hipStreamSynchronize(stream2);
    //matrix_complex_print(Tpowers[8], n);
    printf("-----------> %lf\n", Tpowers[8][0].x );
    


    // ISSUE:
    clock_t norm_begin = clock();
    d4 = pow(calculate_one_norm_New_complex(Tpowers[4], n), (1.0 / 4));
    d6 = pow(calculate_one_norm_New_complex(Tpowers[6], n), (1.0 / 6));
    d8 = pow(calculate_one_norm_New_complex(Tpowers[8], n), (1.0 / 8));
    d10 = pow(calculate_one_norm_New_complex(Tpowers[10], n), (1.0 / 10));
    clock_t norm_end = clock();
    double total_norm_time =  (double)(norm_end - norm_begin) / CLOCKS_PER_SEC;
    printf("TOTAL NORM TIME IS: %lf\n", total_norm_time);
    eta1 = fmax(d4, d6);


    int m_val = 0;

    // We know that we need to calculate ell:


    if (eta1 <= theta[1] && ell(A, temp_new, error_coefficients[1], 3, n, scale_total_time) == 0.0) {
        m_val = 3;

    }
    if (eta1 <= theta[2] && ell(A, temp_new, error_coefficients[2], 5, n, scale_total_time) == 0.0) {
        m_val = 5;

    }


    eta3 = fmax(d6, d8);

    if (eta3 <= theta[3] && ell(A, temp_new, error_coefficients[3], 7, n, scale_total_time) == 0.0) {
        m_val = 7;
    }

    if (eta3 <= theta[4] && ell(A, temp_new, error_coefficients[4], 0, n, scale_total_time) == 0.0) {
        m_val = 9;
    }


    

    eta4 = fmax(d8, d10);
    eta5 = fmin(eta3, eta4);
    
    s = fmax(ceil(log2(eta5 / theta[4])), 0);



    
    hipMemset(d_A, 0, n*n*sizeof(hipDoubleComplex));
    hipMemcpy(d_A, A, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    


    // Kernel is very fast, mem copies lead to longer runtime then using CPU blas matrix scalar
    clock_t scale_begin = clock();
    hipMemcpy(d_A, A, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    scale_tester(handle, d_A, NULL, d_C, make_hipDoubleComplex(1 / pow(2, s), 0), n); // GPU
    hipMemcpy(A, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    //matrix_Scale_New(A, A, make_hipDoubleComplex(1 / pow(2, s), 0), n, scale_total_time); // CPU
    clock_t scale_end = clock();
    double scale_time = (double)(scale_end - scale_begin) / CLOCKS_PER_SEC;
    
    printf("TIME TO SCALE: %lf\n", scale_time);
    
    //hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, int n
    
    printf("the scale is: \n ");
    //matrix_complex_print(A, n);
    exit(0);
    

    s = s + ell(A, temp_new, error_coefficients[4], 13, n, scale_total_time);

    if (isinf(s)) { // Revert to old estimate
       	int exp;
        double t = frexp(calculate_one_norm_New_complex(A, n) / theta[4], &exp);
        s = s - (t == 0.5);
    } else {
        m_val = 13;
    }

    printf("s is: %lf", s);
    if ((int) s != 0) {	// Rescale the matrix powers array
        printf("HAVE TO RESCALE!\n");
    	
    	hipDoubleComplex multiplier = make_hipDoubleComplex(0, 0);

      	multiplier.x = 1.0 / pow(2, (s * 1));
        matrix_Scale_New(Tpowers[1], Tpowers[1], multiplier, n, scale_total_time);
    
        multiplier.x = 1.0 / pow(2, (s * 2));
        matrix_Scale_New(Tpowers[2], Tpowers[2], multiplier, n, scale_total_time);

        multiplier.x = 1.0 / pow(2, (s * 4));
        matrix_Scale_New(Tpowers[4], Tpowers[4], multiplier, n, scale_total_time);

        multiplier.x = 1.0 / pow(2, (s * 6));
        matrix_Scale_New(Tpowers[6], Tpowers[6], multiplier, n, scale_total_time);
     
    }
printf("HERE\n");


   	// PADE APPROXIMATION:
    
    double c[15] = {1};
    
    get_pade_coefficients(c, m_val);
    
    set_Identity_New(identity_new, n);


    if (m_val == 3 || m_val == 5 || m_val == 7 || m_val == 9) {

    	int strt = sizeof(Tpowers) + 2;
        for (int k = strt; k < m_val - 1; k += 2) {
            matrix_Multiply(handle, Tpowers[2], Tpowers[k-2], Tpowers[k], d_A, d_B, d_C, n, multiply_total_time);

        }

        matrix_Scale_New(identity_new, U_new, make_hipDoubleComplex (c[1], 0), n, scale_total_time);
        matrix_Scale_New(identity_new, V_new, make_hipDoubleComplex (c[0], 0), n, scale_total_time);

        for (int j = m_val; j > n; j -= 2) {

            matrix_Scale_New(Tpowers[j - 1], temp_new, make_hipDoubleComplex(c[j + 1], 0), n, scale_total_time);
            matrixAdd_New(U_new, temp_new, U_new, n);

            matrix_Scale_New(Tpowers[j - 1], temp_new, make_hipDoubleComplex(c[j], 0), n, scale_total_time);
            matrixAdd_New(V_new, temp_new, V_new, n);
        }

         matrix_Multiply(handle, U_new, A, temp_new, d_A, d_B, d_C, n, multiply_total_time);
         memcpy(U_new, temp_new, n * n * sizeof(hipDoubleComplex));
     }


    if (m_val == 13) {

        // ------------------ TODO --------------
        // Compute more of the below on the GPU
        // Asses any speed-up for scale & one-norm
        // Attempt to overlap

        // CALCULATE U:

        // overlap: scale 6, scale 4, scale 2, add on gpu, scale 6, scale 4, scale 2, add on gpu

        matrix_Scale_New(Tpowers[6], temp_new, make_hipDoubleComplex(c[13], 0), n, scale_total_time);
   
        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));

        matrixAdd_New(temp_new, temp_2_new, temp_2_new, n);
    
        matrix_Scale_New(Tpowers[4], temp_new, make_hipDoubleComplex(c[11], 0), n, scale_total_time);
        
        matrixAdd_New(temp_new, temp_2_new, temp_2_new, n);

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[2], temp_new, make_hipDoubleComplex(c[9], 0), n, scale_total_time);

        matrixAdd_New(temp_new, temp_2_new, temp_2_new, n);

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));
 
        matrix_Multiply(handle, Tpowers[6], temp_2_new, temp_new, d_A, d_B, d_C, n, multiply_total_time);  

        matrix_Scale_New(Tpowers[6], temp_2_new, make_hipDoubleComplex(c[7], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        matrix_Scale_New(Tpowers[4], temp_2_new, make_hipDoubleComplex(c[5], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        matrix_Scale_New(Tpowers[2], temp_2_new, make_hipDoubleComplex(c[3], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        set_Identity_New(identity_new, n);
     
        matrix_Scale_New(identity_new, temp_2_new, make_hipDoubleComplex(c[1], 0), n, scale_total_time);

        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        memset(U_new, 0, n * n * sizeof(hipDoubleComplex));	// IS THIS NEEDED?

        matrix_Multiply(handle, temp_new, Tpowers[1], U_new, d_A, d_B, d_C, n, multiply_total_time);



        
        // CALCULATE V:

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[6], temp_new, make_hipDoubleComplex(c[12], 0), n, scale_total_time);

        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[4], temp_2_new, make_hipDoubleComplex(c[10], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_2_new, n);

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[2], temp_new, make_hipDoubleComplex(c[8], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_2_new, n);

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));

        matrix_Multiply(handle, temp_2_new, Tpowers[6], temp_new, d_A, d_B, d_C, n, multiply_total_time);

        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[6], temp_2_new, make_hipDoubleComplex(c[6], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[4], temp_2_new, make_hipDoubleComplex(c[4], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));
        matrix_Scale_New(Tpowers[2], temp_2_new, make_hipDoubleComplex(c[2], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, temp_new, n);

        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));
        set_Identity_New(identity_new, n);
        matrix_Scale_New(identity_new, temp_2_new, make_hipDoubleComplex(c[0], 0), n, scale_total_time);
        matrixAdd_New(temp_new, temp_2_new, V_new, n);

        



        // Must wait for above (synchronization point):
        // CALCULATE V-U

        matrix_Subtract_New(V_new, U_new, V_new, n);
       
        matrix_Scale_New(U_new, U_new, make_hipDoubleComplex(2,0), n, scale_total_time);
        //matrix_scale_kernel(handle, U_new, make_hipDoubleComplex(2,0), n);
        memset(temp_2_new, 0, n * n * sizeof(hipDoubleComplex));

        clock_t inverse_begin = clock();
    
        InverseOfMatrix_Alternative_Two(V_new, temp_2_new, n, identity_new);
       
        clock_t inverse_end = clock();
        double inverse_total_time = (double)(inverse_end - inverse_begin) / CLOCKS_PER_SEC;

        memset(temp_new, 0, n * n * sizeof(hipDoubleComplex));

        matrix_Multiply(handle, temp_2_new, U_new, temp_new, d_A, d_B, d_C, n, multiply_total_time);

       	// CALCULATE F:
        matrixAdd_New(temp_new, identity_new, temp_new, n);
        
        // SQUARE THE MATRIX S TIMES - Sequential:
        matrix_Square(handle, temp_new, temp_2_new, d_A, d_C, n, multiply_total_time, s);

        // PERFORMANCE OUTPUT:
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;	// End recording expm execution
        
        printf("\n----------------------- MATRIX OPERATIONS PERCENTAGE BREAKDOWN -----------------------\n");
        printf("\n TOTAL TIME ELAPSED: %lf seconds \n", time_spent);
        printf("\n INVERSE: %lf%% \n", (inverse_total_time/time_spent)*100);
        printf("\n SCALE: %lf%% \n", (scale_total_time[0]/time_spent)*100);
        printf("\n MULTIPLY: %lf%% \n\n", (multiply_total_time[0]/time_spent)*100);

     }


     printf("----> %lf\n", temp_2_new[0].x );
    // Free host memory
    hipHostFree(identity_new);
    hipHostFree(U_new);
    hipHostFree(V_new);
    hipHostFree(temp_new);
    hipHostFree(temp_2_new);
    free(scale_total_time);
    free(multiply_total_time);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

// Resources
// http://www.netlib.org/utk/people/JackDongarra/PAPERS/Factor_Inversion_Million_Matrices-iccs17.pdf
// http://mathforcollege.com/nm/mws/che/04sle/mws_che_sle_spe_luinverse.pdf