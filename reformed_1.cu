#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <string>
#include <stdio.h>  
#include <stdlib.h>
#include <float.h>
#include <memory.h>
#include <math.h>
#include <stdbool.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cblas.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>
#include "expm.h"

// *** Matrix exponential program acting on matrices of type : [Double Complex] - 10/03/2019 ***

// Converting synchronous program to be more asynchronous and thus allow for better utilization of GPU resources
// and GPU memory copies as to reduce overall runtime
// Approaches for improvement:
// [1] Overlap of memory copies -> The hardware allows for concurrent memory copies that are performed in opposing
// directions.
// [2] Overlap of GPU kernels (computation) -> THe hardware allows for some concurrency of kernel exeution however
// the amount of overlap witnessed depends on how many GPU resources the kernels use, once there are suffcient resources
// released from one kernel, the other kernels execution may begin.
// [3] Need to be aware of data dependencies, this involves ensuring that dependent (synchronous in nature) kernels are
// assigned to the same stream or are seperated by a synchronisation point as to ensure that correct results are achieved.
// [4] Reducing memory copies by keeping data on the device as much as possible, this involves converting current CPU functions
// into GPU functions that will give the same of better runtime (when run on a better GPU) such that less data transfer is
// required.
// [5] Not yet looked at, but another possibility to reduce the number of memcopies but stillm copy the same total amount
// of data is to look at combining arrays representing matrices into a larger memory space and then use offsets to refer to 
// the matrix of interest (perhaps better banwidth utilisation but perhaps less readable).


// TIME BREAKDOWN BY COMPUTE OPERATION - FROM PROFILER:
// 98.4% of runtime is spent in the multiply kernel
// 1.6% of runtime is spent in the inverse kernels

// Memory copies take small relative to operation time to complete for larger matrices
// Small overlap between dgemm kernels due to using large number of available resources so only a small
// amount of kernel concurrency possible
// For smaller matrices (e.g. 50 * 50) more concurrency is witnissed (50% overlap) - 'A more concurrent
// execution.'


void matrix_complex_print(hipDoubleComplex* A, int network_size){
	for (int j = 0; j < network_size; j++){
		printf("[");
		for (int k = 0; k < network_size; k++){
			printf(" %.15lf ", A[(j*network_size) + k].x );
			printf("+");
			printf(" %.15lfi ", A[(j*network_size) + k].y );
		}
		printf("]");
		printf("\n");
	}
}

void matrix_Square_Reduced(hipblasHandle_t &handle, hipDoubleComplex *d_A, hipDoubleComplex *d_C, int n, double* multiply_total_time, int s){
    
    clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;
    
    for (int k = 0; k < s; k++) {
        cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_A, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication
        hipMemcpy(d_A, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    }
   
    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;


}



void matrix_Square(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex *d_C, int n, double* multiply_total_time, int s){
	
    clock_t multiply_begin = clock();

	const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;
	
	hipMemcpy(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice);	// Copy first operand to the device (only one copy is needed for the squaring phase)

	for (int k = 0; k < s; k++) {
		cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_A, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication
        hipMemcpy(d_A, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);	// Copy product back to host
 	
 	clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;

}



void matrix_Multiply_Reduced(hipblasHandle_t &handle, hipDoubleComplex *d_A, hipDoubleComplex* d_B, hipDoubleComplex *d_C, int n, double* multiply_total_time){

    clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_B, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication

    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;


}


void matrix_Multiply(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex* d_B, hipDoubleComplex *d_C, int n, double* multiply_total_time){

    clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    hipMemcpy(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice);	// Copy first operand to the device
    hipMemcpy(d_B, B, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice); // Copy second operand to the device
    
    cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_B, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication

    hipMemcpy(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);	// Copy product back to host
    
    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;
}


void matrix_Multiply_With_Streams(hipblasHandle_t &handle, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, hipDoubleComplex *d_A, hipDoubleComplex* d_B, hipDoubleComplex *d_C, int n, double* multiply_total_time, hipStream_t stream){
 clock_t multiply_begin = clock();

    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    hipMemcpyAsync(d_A, A, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice, stream); // Copy first operand to the device
    hipMemcpyAsync(d_B, B, n * n * sizeof(hipDoubleComplex),hipMemcpyHostToDevice, stream); // Copy second operand to the device

    hipblasSetStream(handle, stream);
    cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_B, n, d_A, n, beta, d_C, n); // Perform the cublas matrix multiplication

    hipMemcpyAsync(C, d_C, n * n * sizeof(hipDoubleComplex),hipMemcpyDeviceToHost, stream); // Copy product back to host
    
    clock_t multiply_end = clock();
    double time_spent = (double)(multiply_end - multiply_begin) / CLOCKS_PER_SEC;
    multiply_total_time[0] = multiply_total_time[0] + time_spent;

}


hipDoubleComplex **get_Matrix_Powers_New(hipDoubleComplex *A, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, hipblasHandle_t handle, int n, double* multiply_total_time) {

    hipDoubleComplex **Tpowers = (hipDoubleComplex **) malloc(11 * sizeof(hipDoubleComplex *));

    for (int i = 0; i < 11; i++) {
        Tpowers[i] = (hipDoubleComplex *) malloc(n * n * sizeof(hipDoubleComplex));
    }

    memcpy(Tpowers[1], A, n * n * sizeof(hipDoubleComplex));
    

    // To calculate Tpoers[2] & Tpowers[4] this is sequential
    matrix_Multiply(handle, Tpowers[1], Tpowers[1], Tpowers[2], d_A, d_B, d_C, n, multiply_total_time);
     
    matrix_Multiply(handle, Tpowers[2], Tpowers[2], Tpowers[4], d_A, d_B, d_C, n, multiply_total_time);



    matrix_Multiply(handle, Tpowers[4], Tpowers[2], Tpowers[6], d_A, d_B, d_C, n, multiply_total_time);
   
    matrix_Multiply(handle, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time);

    return Tpowers;
}



// *** CURRENT WORK ***
// Need description of the method being used
// Note that this memthod calls the DGEMM multiplication kernel and as so the functions suffers from the poor FP64
// performance see for the dtandard CUBLAS DGEMM function when ran on the currenctly used GEFORCE GTX 960

/* Calculate matrix inverse through LU factorisation -> AX = I for an input matrix A, the identity matrix I and the inverse matrix X
 [1]
 * The LU factorization of the input double complex matrix is computed using the CUSOLVER function hipsolverDnZgetrf
 * This factors a matrix as the product of a lower triangular and upper triangular marix -> [A = LU] 
 
 [2]
 * Then the resultant linear system is solved:
 * A^-1 = U^-1 * L^-1, hence need to invert each of the two matrices
 * As A*A^-1 = I, then L*U*A^-1 = I and we know L, U and I
 * We can the solve a set of linear equations to find the inverse using hipsolverDnZgetrs

 * SOURCE: https://math.stackexchange.com/questions/1009916/easy-way-to-calculate-inverse-of-an-lu-decomposition
*/
void InverseOfMatrix_Alternative_Two(hipDoubleComplex* d_in, hipDoubleComplex* d_out, int n){ 
    
    hipsolverStatus_t status;	// Link to the cusolver context
    hipsolverHandle_t handler;
    status = hipsolverDnCreate(&handler);

    int* dLUPivots_ALT;
    int* dLUInfo_ALT;
    hipDoubleComplex *buffer = NULL;
    int bufferSize = 0;
    int h_info = 0;
     
    hipMalloc(&dLUPivots_ALT, n * sizeof(int)), "Failed to allocate dLUPivots!";
    hipMalloc(&dLUInfo_ALT, sizeof(int)), "Failed to allocate dLUInfo!";

    hipsolverDnZgetrf_bufferSize(handler, n, n, (hipDoubleComplex*)d_in, n, &bufferSize);
    hipMalloc(&buffer, sizeof(hipDoubleComplex)*bufferSize);
  
    status = hipsolverDnZgetrf(handler, n, n, d_in, n, buffer, dLUPivots_ALT, dLUInfo_ALT);
    if(status!=HIPSOLVER_STATUS_SUCCESS){
        printf("ERROR!!\n");
    } 

    hipMemcpy(&h_info, dLUInfo_ALT, sizeof(int), hipMemcpyDeviceToHost);
 
    if ( h_info != 0 ){
        fprintf(stderr, "Error: LU factorization failed\n");
        printf("%d\n", h_info );
    }
      
    status = hipsolverDnZgetrs(handler, HIPBLAS_OP_N, n, n, d_in, n, dLUPivots_ALT, d_out, n, dLUInfo_ALT);
    hipDeviceSynchronize();
     if(status!=HIPSOLVER_STATUS_SUCCESS){
        printf("ERROR!!\n");
    } 
    hipMemcpy(&h_info, dLUInfo_ALT, sizeof(int), hipMemcpyDeviceToHost);
        if ( h_info != 0 ){
        fprintf(stderr, "Error: LU factorization failed\n");
    }

    // Free device memory:
    hipFree(dLUPivots_ALT);
    hipFree(dLUInfo_ALT);
    hipFree(buffer);
}

void matrix_Subtract_New(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n) { // PARALLEL CANDIDATE

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            c[(n * i) + j] = hipCsub(a[(n * i) + j], b[(n * i) + j]); // Complex subtraction
        }
    }
}


void matrixAdd_New(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n) { // PARALLEL CANDIDATE

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            c[(n * i) + j] = hipCadd(a[(n * i) + j], b[(n * i) + j]); // Complex addition
        }
    }
}

void matrix_add_Tester(hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, int n){
    
    //const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    //hipblasZdgmm(handle, mode, n,n, d_A, n, d_X, n, d_C, n);
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, alpha, d_A, n, beta, d_B, n, d_C, n);
}

void matrix_subtract_Tester(hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, int n){
    
    //const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(-1, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    //hipblasZdgmm(handle, mode, n,n, d_A, n, d_X, n, d_C, n);
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, alpha, d_A, n, beta, d_B, n, d_C, n);
}

void set_Identity_New(hipDoubleComplex *i_matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                i_matrix[(n * i) + j].x = 1;
            } else {
                i_matrix[(n * i) + j].x = 0;
            }
        }
    }
}



void scale_tester(hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, const hipDoubleComplex alf, int n, double* scale_total_time ){

    clock_t scale_begin = clock();
    //const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    //hipblasZdgmm(handle, mode, n,n, d_A, n, d_X, n, d_C, n);
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, alpha, d_A, n, beta, d_B, n, d_C, n);

    clock_t scale_end = clock();
    double time_spent = (double)(scale_end - scale_begin) / CLOCKS_PER_SEC;
    scale_total_time[0] = scale_total_time[0] + time_spent;
}



void scale_tester_alt(hipblasHandle_t handle, hipDoubleComplex* d_A, hipDoubleComplex* d_B, hipDoubleComplex* d_C, const hipDoubleComplex alf, int n, double* scale_total_time ){

    clock_t scale_begin = clock();
    //const hipDoubleComplex alf = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex bet = make_hipDoubleComplex(1, 0);
    const hipDoubleComplex *alpha = &alf;
    const hipDoubleComplex *beta = &bet;

    //hipblasZdgmm(handle, mode, n,n, d_A, n, d_X, n, d_C, n);
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, alpha, d_A, n, beta, d_B, n, d_C, n);

    clock_t scale_end = clock();
    double time_spent = (double)(scale_end - scale_begin) / CLOCKS_PER_SEC;
    scale_total_time[0] = scale_total_time[0] + time_spent;
}


void matrix_scale_kernel(hipblasHandle_t &handle, hipDoubleComplex* A, hipDoubleComplex scale, int n){

	const hipDoubleComplex alf = make_hipDoubleComplex(scale.x, scale.y);
  
    const hipDoubleComplex *alpha = &alf;

	hipblasZscal(handle,n*n,alpha,A,1);
}

void matrix_Scale_New(hipDoubleComplex *a, hipDoubleComplex *scaled, hipDoubleComplex scale, int n, double* scale_total_time ) {
    clock_t scale_begin = clock();

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            //scaled[(n * i) + j] = a[(n * i) + j] * scale;
            scaled[(n * i) + j] = hipCmul(a[(n * i) + j],scale); // Complex multiplication
        }
    }

    clock_t scale_end = clock();
    double time_spent = (double)(scale_end - scale_begin) / CLOCKS_PER_SEC;
    scale_total_time[0] = scale_total_time[0] + time_spent;
}


void matrix_Absolute_New(hipDoubleComplex *a, hipDoubleComplex *b, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            b[(n * i) + j].x = hipCabs((a[(n * i) + j]));
            b[(n * i) + j].y = 0;
        }
    }
}

// NEED TO FIND A GPU EQUIVELANT -> NOT WORTHIT DUE TO ONE OP PER DATA ITEM AND LARGE MEMCOPY COSTS.
double calculate_one_norm_New_complex(const hipDoubleComplex *A, int n) {
    double max = -DBL_MAX;
	double count;
    for (int i = 0; i < n; i++) {
        count = 0;
        for (int j = 0; j < n; j++) {
            count += hipCabs((A[(n * j) + i]));
        }
        if (count > max) {;
            max = count;
        };
    }
    return max;
}


// COMPUTING OPTIMAL PARAMETERS
double ell(hipDoubleComplex *A, hipDoubleComplex *temp_new, hipDoubleComplex *d_A, double coeff, int m_val, int n, double* scale_total_time, hipblasHandle_t &handle, hipStream_t stream) {

    double norm_one, norm_two, p, alpha, output;
    memcpy(A, temp_new, n * n * sizeof(hipDoubleComplex));
    
    matrix_Absolute_New(A, temp_new, n);

    p = pow(coeff, (1.0 / (2 * m_val + 1)));
    
    hipMemcpyAsync(d_A, temp_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    scale_tester(handle, d_A, NULL, d_A, make_hipDoubleComplex(p, 0), n, scale_total_time);
    hipMemcpyAsync(temp_new, d_A, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);
    
    norm_one = calculate_one_norm_New_complex(A, n); // Overlap GPU & CPU WORK WITH ASYNC CALLS

    hipDeviceSynchronize();
    norm_two = calculate_one_norm_New_complex(temp_new, n);
    
    alpha = norm_two / norm_one;

    output = fmax(ceil(log2((2 * alpha) / 2.220446049250313e-16) / (2 * m_val)), 0);

    return output;
}

void get_pade_coefficients(double *buf, int m) {

    double coefficients[5][14] = {
            {120, 60, 12, 1},
            {30240, 15120, 3360, 420, 30, 1},
            {17297280, 8648640, 1995840, 277200, 25200, 1512, 56 ,1},
            {17643225600, 8821612800, 2075673600, 302702400, 30270240, 2162160, 110880, 3960, 90, 1},
            {64764752532480000, 32382376266240000, 7771770303897600, 1187353796428800, 129060195264000, 10559470521600, 670442572800, 33522128640, 1323241920, 40840800, 960960, 16380, 182, 1}
        };

    switch (m) {

        case 3  : {
            buf = coefficients[0];

        }
        case 5  : {
            buf = coefficients[1];
        }
        case 7  : {
            buf = coefficients[2];
        }

        case 9  : {
            buf = coefficients[3];
        }
        case 13  : {
            for (int i = 0; i < sizeof(coefficients[4]) / sizeof(double); i++) {
                buf[i] = coefficients[4][i];
            }
        }
        default:
            break;
    }
}

int main(){

    hipDoubleComplex* A;
    int n = 1024;

    // Allocate the pinned memory:
    hipHostMalloc((void**)&A, n*n*sizeof(hipDoubleComplex));


    // Initialize the pinned memory:
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; ++j)
        {
           A[(n*i) + j].x = 0.01;
           A[(n*i) + j].y = 0.0035;
        }
    }
    

    clock_t setup_begin = clock();
    // VARIABLES TO HOLD TOTAL COMPONENT TIMES:
    double* scale_total_time = (double *) malloc(1 * sizeof(double));
    double* multiply_total_time = (double *) malloc(1 * sizeof(double));

    clock_t begin = clock();  // Begin recording expm execution

    // CUBLAS HANDLE:
    hipblasHandle_t handle, handle2, handle3, handle4;
    hipblasCreate(&handle);
    hipblasCreate(&handle2);
    hipblasCreate(&handle3);
    hipblasCreate(&handle4);

    // Allocate 3 arrays on GPU
    hipDoubleComplex *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(hipDoubleComplex));
    hipMalloc(&d_B, n * n * sizeof(hipDoubleComplex));
    hipMalloc(&d_C, n * n * sizeof(hipDoubleComplex));


    double theta[5] = {1.495585217958292e-002, 2.539398330063230e-001,
                       9.504178996162932e-001, 2.097847961257068e+000,
                       5.371920351148152e+000};

    double error_coefficients[5] = {1 / 100800.0, 1 / 10059033600.0, 1 / 4487938430976000.0,
                                    1 / 113250775606021113483283660800000000.0,
                                    1 / 113250775606021113483283660800000000.0};

    // Allocate temporary arrays to hold temporary matrices used at various stages in the calculation
    hipDoubleComplex *identity_new;
    hipDoubleComplex *U_new;
    hipDoubleComplex *V_new;
    hipDoubleComplex *temp_new;
    hipDoubleComplex *temp_2_new;
    hipHostMalloc((void**)&identity_new, n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&U_new, n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&V_new, n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&temp_new, n * n * sizeof(hipDoubleComplex));
    hipHostMalloc((void**)&temp_2_new, n * n * sizeof(hipDoubleComplex));
    

    // Create cuda streams to enable asynchronous behaviour:
    hipStream_t stream2, stream3, stream4;
    
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    float* A_d;
    hipMalloc(&A_d, n*n*sizeof(hipDoubleComplex));

    double d4, d6, d8, d10, eta1, eta3, eta4, eta5, s;

    hipDoubleComplex **Tpowers;
    hipHostMalloc((void**)&Tpowers, 11 * sizeof(hipDoubleComplex*));

    for (int i = 0; i < 11; i++) {
        hipHostMalloc((void**)&Tpowers[i], n *n* sizeof(hipDoubleComplex));
    }

    clock_t setup_end = clock();
    double total_setup_time =  (double)(setup_end - setup_begin) / CLOCKS_PER_SEC;

    memcpy(Tpowers[1], A, n * n * sizeof(hipDoubleComplex));

    // To calculate Tpowers[2] & Tpowers[4] this is sequential
    matrix_Multiply(handle, Tpowers[1], Tpowers[1], Tpowers[2], d_A, d_B, d_C, n, multiply_total_time);
    matrix_Multiply(handle, Tpowers[2], Tpowers[2], Tpowers[4], d_A, d_B, d_C, n, multiply_total_time);

    // Attempt to have concurrency:
    //matrix_Multiply_With_Streams(handle, Tpowers[4], Tpowers[2], Tpowers[6], d_A, d_B, d_C, n, multiply_total_time, stream1);
    //matrix_Multiply_With_Streams(handle2, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time, stream2);
    matrix_Multiply(handle, Tpowers[4], Tpowers[2], Tpowers[6], d_A, d_B, d_C, n, multiply_total_time);
    matrix_Multiply(handle, Tpowers[4], Tpowers[4], Tpowers[8], d_A, d_B, d_C, n, multiply_total_time);


    // ISSUE:
    clock_t norm_begin = clock();
    d4 = pow(calculate_one_norm_New_complex(Tpowers[4], n), (1.0 / 4));
    d6 = pow(calculate_one_norm_New_complex(Tpowers[6], n), (1.0 / 6));
    d8 = pow(calculate_one_norm_New_complex(Tpowers[8], n), (1.0 / 8));
    d10 = pow(calculate_one_norm_New_complex(Tpowers[10], n), (1.0 / 10));
    clock_t norm_end = clock();
    double total_norm_time =  (double)(norm_end - norm_begin) / CLOCKS_PER_SEC;
    eta1 = fmax(d4, d6);


    int m_val = 0;
    hipblasSetStream(handle2, stream2);

    // We know that we need to calculate ell:


    if (eta1 <= theta[1] && ell(A, temp_new, d_A, error_coefficients[1], 3, n, scale_total_time, handle2, stream2) == 0.0) {
        m_val = 3;

    }
    if (eta1 <= theta[2] && ell(A, temp_new, d_A, error_coefficients[2], 5, n, scale_total_time, handle2, stream2) == 0.0) {
        m_val = 5;

    }

    eta3 = fmax(d6, d8);

    if (eta3 <= theta[3] && ell(A, temp_new, d_A, error_coefficients[3], 7, n, scale_total_time, handle2, stream2) == 0.0) {
        m_val = 7;
    }

    if (eta3 <= theta[4] && ell(A, temp_new, d_A, error_coefficients[4], 0, n, scale_total_time, handle2, stream2) == 0.0) {
        m_val = 9;
    }


    eta4 = fmax(d8, d10);
    eta5 = fmin(eta3, eta4);
    
    s = fmax(ceil(log2(eta5 / theta[4])), 0);
    
    hipMemset(d_A, 0, n*n*sizeof(hipDoubleComplex));
    hipMemcpy(d_A, A, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // Kernel is very fast, mem copies lead to longer runtime then using CPU cblas matrix scalar
 
    hipMemcpy(d_A, A, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    scale_tester(handle, d_A, NULL, d_C, make_hipDoubleComplex(1 / pow(2, s), 0), n, scale_total_time);
    hipMemcpy(A, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    s = s + ell(A, temp_new, d_A, error_coefficients[4], 13, n, scale_total_time,  handle2, stream2);

    // Attempt to calculate s without leaving the GPU

    if (isinf(s)) { // Revert to old estimate
       	int exp;
        double t = frexp(calculate_one_norm_New_complex(A, n) / theta[4], &exp);
        s = s - (t == 0.5);
    } else {
        m_val = 13;
    }

    hipblasSetStream(handle2, stream2);
    hipblasSetStream(handle3, stream3);
    hipblasSetStream(handle4, stream4);
    
    if ((int) s != 0) {	// Rescale the matrix powers array
    	      
        // Independent data, Work is streamified:
        hipMemcpyAsync(d_A, Tpowers[1], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester(handle2, d_A, NULL, d_A, make_hipDoubleComplex(1.0 / pow(2, (s * 1)), 0), n, scale_total_time);
        hipMemcpyAsync(Tpowers[1], d_A, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream2);
    
        hipMemcpyAsync(d_B, Tpowers[2], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester(handle3, d_B, NULL, d_B, make_hipDoubleComplex(1.0 / pow(2, (s * 2)), 0) , n, scale_total_time);
        hipMemcpyAsync(Tpowers[2], d_B, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream3);

        hipMemcpyAsync(d_C, Tpowers[4], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream4);
        scale_tester(handle4, d_C, NULL, d_C, make_hipDoubleComplex(1.0 / pow(2, (s * 4)), 0), n, scale_total_time);
        hipMemcpyAsync(Tpowers[4], d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream4);

        matrix_Scale_New(Tpowers[6], Tpowers[6], make_hipDoubleComplex(1.0 / pow(2, (s * 6)), 0), n, scale_total_time);

        hipDeviceSynchronize();

    }

   	// PADE APPROXIMATION:
    
    double c[15] = {1};
    
    get_pade_coefficients(c, m_val);
    
    set_Identity_New(identity_new, n);
    if (m_val == 3 || m_val == 5 || m_val == 7 || m_val == 9) {

    	int strt = sizeof(Tpowers) + 2;
        for (int k = strt; k < m_val - 1; k += 2) {
            matrix_Multiply(handle, Tpowers[2], Tpowers[k-2], Tpowers[k], d_A, d_B, d_C, n, multiply_total_time);

        }

        matrix_Scale_New(identity_new, U_new, make_hipDoubleComplex (c[1], 0), n, scale_total_time);
        matrix_Scale_New(identity_new, V_new, make_hipDoubleComplex (c[0], 0), n, scale_total_time);

        for (int j = m_val; j > n; j -= 2) {

            matrix_Scale_New(Tpowers[j - 1], temp_new, make_hipDoubleComplex(c[j + 1], 0), n, scale_total_time);
            matrixAdd_New(U_new, temp_new, U_new, n);

            matrix_Scale_New(Tpowers[j - 1], temp_new, make_hipDoubleComplex(c[j], 0), n, scale_total_time);
            matrixAdd_New(V_new, temp_new, V_new, n);
        }

         matrix_Multiply(handle, U_new, A, temp_new, d_A, d_B, d_C, n, multiply_total_time);
         memcpy(U_new, temp_new, n * n * sizeof(hipDoubleComplex));
     }


    // TODO: Look at storing Tpowers on the GPU
    // Remove extra cudaDeviceSync
    // Keep result on GPU without copy after the multiplication

    if (m_val == 13) {
        
        // Bind the CUDA streams to CUBLAS handles for asynchronous work:
        hipblasSetStream(handle2, stream2);
        hipblasSetStream(handle3, stream3);

        // CALCULATE U:
        hipMemcpyAsync(d_A, Tpowers[6], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester(handle3, d_A, NULL, d_C, make_hipDoubleComplex(c[13], 0), n, scale_total_time); // GPU

        hipMemcpyAsync(d_B, Tpowers[4], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_B, d_C, d_C,  make_hipDoubleComplex(c[11], 0), n, scale_total_time);

        hipMemcpyAsync(d_A, Tpowers[2], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester_alt(handle3, d_A, d_C, d_C,  make_hipDoubleComplex(c[9], 0), n, scale_total_time);
        
        // SYNCHRONIZATION POINT:
        hipMemcpy(temp_2_new, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);   
        matrix_Multiply(handle, Tpowers[6], temp_2_new, temp_new, d_A, d_B, d_C, n, multiply_total_time);

        hipMemcpyAsync(d_A, Tpowers[6], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_A, d_C, d_C,  make_hipDoubleComplex(c[7], 0), n, scale_total_time);
        
        hipMemcpyAsync(d_B, Tpowers[4], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester_alt(handle3, d_B, d_C, d_C,  make_hipDoubleComplex(c[5], 0), n, scale_total_time);

        hipMemcpyAsync(d_A, Tpowers[2], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_A, d_C, d_C,  make_hipDoubleComplex(c[3], 0), n, scale_total_time);
    
        // SYNCHRONIZATION POINT:
        set_Identity_New(identity_new, n); 
        hipMemcpy(d_A, identity_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        scale_tester_alt(handle, d_A, d_C, d_C,  make_hipDoubleComplex(c[1], 0), n, scale_total_time);
        hipMemcpyAsync(d_B, Tpowers[1], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        matrix_Multiply_Reduced(handle, d_C, d_B, d_A, n, multiply_total_time);
        hipMemcpy(U_new, d_A, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    
        // CALCULATE V:
        hipMemcpyAsync(d_A, Tpowers[6], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d_B, Tpowers[4], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester(handle2, d_A, NULL, d_A,  make_hipDoubleComplex(c[12], 0), n, scale_total_time);
        scale_tester(handle3, d_B, NULL, d_B,  make_hipDoubleComplex(c[10], 0), n, scale_total_time);
        hipDeviceSynchronize();
        matrix_add_Tester(handle, d_A, d_B, d_B, n);
        
        hipMemcpyAsync(d_C, Tpowers[2], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_C, d_B, d_C,  make_hipDoubleComplex(c[8], 0), n, scale_total_time);

        // SYNCHRONIZATION POINT:
        hipMemcpyAsync(d_A, Tpowers[6], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        hipMemcpy(d_B, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
        matrix_Multiply_Reduced(handle, d_A, d_B, d_C, n, multiply_total_time);
        hipMemcpy(temp_new, d_B, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

        hipMemcpyAsync(d_A, Tpowers[6], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_A, d_C, d_C,  make_hipDoubleComplex(c[6], 0), n, scale_total_time);

        hipMemcpyAsync(d_B, Tpowers[4], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream3);
        scale_tester_alt(handle3, d_B, d_C, d_C,  make_hipDoubleComplex(c[4], 0), n, scale_total_time);
        
        hipMemcpyAsync(d_A, Tpowers[2], n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream2);
        scale_tester_alt(handle2, d_A, d_C, d_C,  make_hipDoubleComplex(c[2], 0), n, scale_total_time);
        
        // SYNCHRONIZATION POINT:
        set_Identity_New(identity_new, n);
        hipMemcpy(d_A, identity_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);  
        scale_tester_alt(handle, d_A, d_C, d_C,  make_hipDoubleComplex(c[0], 0), n, scale_total_time);
        hipMemcpy(V_new, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // CALCULATE V-U:
        hipMemcpy(d_A, U_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice); 
        matrix_subtract_Tester(handle, d_C, d_A, d_B, n);
        hipMemcpy(V_new, d_B, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // CALCULATE F:
        scale_tester(handle, d_A, NULL, d_A,  make_hipDoubleComplex(c[0], 0), n, scale_total_time);
        hipMemcpy(U_new, d_A, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

        clock_t inverse_begin = clock();
        hipMemcpy(d_C, identity_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        InverseOfMatrix_Alternative_Two(d_B, d_C, n);
        hipMemcpy(temp_2_new, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

        clock_t inverse_end = clock();
        double inverse_total_time = (double)(inverse_end - inverse_begin) / CLOCKS_PER_SEC;

        matrix_Multiply_Reduced(handle, d_C, d_A, d_B, n, multiply_total_time);
        hipMemcpy(d_A, identity_new, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        matrix_add_Tester(handle, d_B, d_A, d_C, n);  
        hipMemcpy(temp_new, d_C, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
 
        // SQUARE MATRIX F, S TIMES:
        clock_t square_begin = clock();
        matrix_Square_Reduced(handle, d_C, d_A, n, multiply_total_time, s);
        hipMemcpy(temp_2_new, d_A, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        clock_t square_end = clock();
        double square_total_time = (double)(square_end - square_begin) / CLOCKS_PER_SEC;

        // PERFORMANCE OUTPUT:
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;	// End recording expm execution

        printf("******************\n");
        printf("%.15lf\n", temp_2_new[22].x);
      
        printf("\n----------------------- MATRIX OPERATIONS PERCENTAGE BREAKDOWN -----------------------\n");
        printf("\n TOTAL TIME ELAPSED: %lf seconds \n", time_spent);
        printf("\n SETUP: %lf%% \n", (total_setup_time/time_spent)*100);
        printf("\n NORM [CPU]: %lf%% \n", (total_norm_time/time_spent)*100);
        printf("\n INVERSE: %lf%% \n", (inverse_total_time/time_spent)*100);
        printf("\n SCALE: %lf%% \n", (scale_total_time[0]/time_spent)*100);
        printf("\n MULTIPLY: %lf%% \n", (multiply_total_time[0]/time_spent)*100);
        printf("\n SQUARE: %lf%% \n\n", (square_total_time/time_spent)*100);

    }

    // Free host memory
    hipHostFree(identity_new);
    hipHostFree(U_new);
    hipHostFree(V_new);
    hipHostFree(temp_new);
    hipHostFree(temp_2_new);
    free(scale_total_time);
    free(multiply_total_time);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

// Resources
// http://www.netlib.org/utk/people/JackDongarra/PAPERS/Factor_Inversion_Million_Matrices-iccs17.pdf
// http://mathforcollege.com/nm/mws/che/04sle/mws_che_sle_spe_luinverse.pdf