
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_complex.h>

// Dummy C function that takes the input matrix:
extern "C" void expm_initialization(const void * input, void * output, int dim) {
	const hipDoubleComplex * A = (hipDoubleComplex *) input; // Cast void pointers to complex doubles
    hipDoubleComplex * mat_expm = (hipDoubleComplex *) output; // Cast void pointers to complex doubles
    
    for (int i = 0; i < dim * dim; ++i) {
        mat_expm[i] = make_hipDoubleComplex(i, 100); // Peform the expm algorithm
    }
}